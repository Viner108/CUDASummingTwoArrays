#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float *dB, float *dC,int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//���� �� ����� ����� ���� ������������ �� ����� ���� ���������� ����� �� ������ ���������� � ������������ ������ ������
	//��������� ������� i < size
	if (i < size) dC[i] = dA[i] + dB[i];
}