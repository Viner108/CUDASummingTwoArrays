#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float *dB, float *dC,int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//if the number of threads was arbitrary, then the threads of the last block could not access an invalid memory cell
	//added condition i < size
	if (i < size) dC[i] = dA[i] + dB[i];
}