#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h"

int main(int argc, char* argv[])
{
	//���������� ��� ��������� ������� ���������� ������� �� GPU � �� CPU
	float timerValueGPU, timerValueCPU;
	// �������� ������� ��� ������ � ����� ������ �������
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *hA;
	float *dA;
	float *hB;
	float *dB;
	float *hC;
	float *dC;

	// ��������� ������
	int size = 512 * 50000;
	// ����� �����
	int  N_thread = 512;
	// ����� ������
	int N_blocks;
	int i;
	// ���������� ���������� ������
	unsigned int mem_size = sizeof(float) * size;

	hA = (float*)malloc(mem_size);
	hB = (float*)malloc(mem_size);
	hC = (float*)malloc(mem_size);
	// hA, hB, hC ����� � ����������� ������ CPU

	hipError_t err;

	err = hipMalloc((void**)&dA, mem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dB, mem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dC, mem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}
	//dA, dB, dC � ���������� ������ GPU

	for ( i = 0; i < size; i++)
	{
		hA[i] = 1.0f / ((i + 1.0F) * (i + 1.0f));
		hB[i] = expf(1.0f / (1 + 1.0f));
		hC[i] = 0.0f;
	}

	if ((size % N_thread) == 0) {
		N_blocks = size / N_thread;
	}
	else {
		N_blocks = (int)(size / N_thread) + 1;
	}
	dim3 blocks(N_blocks);

	//����� ������� �������
	hipEventRecord(start, 0);
	
	err = hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot copy data host/device : %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot copy data host/device : %s\n", hipGetErrorString(err));
		return 1;
	}
	//hA, hB ���������� � dA, dB �� ����������� ������ � ����������

	function << < N_blocks,N_thread >> > (dA,dB,dC,size);
	
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot copy data device/host : %s\n", hipGetErrorString(err));
		return 1;
	}
	//dC ���������� � hC � ���������� ������ � �����������
	 
	//����� ������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	// ������� � ������������ �������� ������ �� CPU
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);
	//����� ������� �������
	hipEventRecord(start, 0);

	for ( i = 0; i < size; i++)
	{
		hC[i] = hA[i] + hB[i];
	}
	// hA, hB, hC ����� � ����������� ������ CPU
	
	//����� ������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	// ������� � ������������ �������� ������ �� CPU
	hipEventElapsedTime(&timerValueCPU, start, stop);
	printf("\n CPU calculation time: %f ms\n", timerValueCPU);

	// �� ������� ��� ������� GPU ������������ ������ ��� CPU
	printf("\n Rate: %f x\n", timerValueCPU / timerValueGPU);

	free(hA);
	free(hB);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}
